
#include <hip/hip_runtime.h>
#include <cmath>
#include <fstream>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <vector>

using namespace std;

struct Point3D
{
    double x, y, z;
};

struct Triangle
{
    Point3D p1;
    Point3D p2;
    Point3D p3;
};

__device__ 
Point3D difference(Point3D a, Point3D b)
{
    return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__device__ 
Point3D crossProduct(Point3D &v1, Point3D &v2)
{
    return {v1.y * v2.z - v1.z * v2.y, v1.z * v2.x - v1.x * v2.z,
            v1.x * v2.y - v1.y * v2.x};
}

__device__ 
double dotProduct(Point3D &v1, Point3D &v2)
{
    return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

__device__ 
bool rayIntersectsTriangle(
	Point3D rayOrigin, 
	Point3D rayVector,
	Triangle inTriangle
){
    const float EPSILON = 0.0000001;
    Point3D vertex0 = inTriangle.p1;
    Point3D vertex1 = inTriangle.p2;
    Point3D vertex2 = inTriangle.p3;

    Point3D edge1, edge2, h, s, q;
    double a, f, u, v;

    edge1 = difference(vertex1, vertex0);
    edge2 = difference(vertex2, vertex0);

    h = crossProduct(rayVector, edge2);
    a = dotProduct(edge1, h);

    if (a > -EPSILON && a < EPSILON)
        return false;

    f = 1.0 / a;
    s = difference(rayOrigin, vertex0);
    u = f * dotProduct(s, h);

    if (u < 0.0 || u > 1.0)
        return false;

    q = crossProduct(s, edge1);
    v = f * dotProduct(rayVector, q);
    if (v < 0.0 || u + v > 1.0)
        return false;

    double t = f * dotProduct(edge2, q);

    if (t > EPSILON)
        return true;
    return false;
}

__global__
void rayIntersectsAnyTrianglesKernel(
	Point3D rayOrigin, 
	Point3D *ps,
	Triangle *ts,
	int Nt,
	bool *result
){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	Point3D point = ps[idx];
	
	Point3D dir = point;
		
	result[idx] = false;
	
	for(int i = 0; i < Nt; i++){
		Triangle t = ts[i];
		
		bool r = rayIntersectsTriangle(rayOrigin, dir, t);
		if(r){
			result[idx] = true;
		}
	}	
}

vector<Point3D> readPoints(string filename)
{
    vector<Point3D> punti;
    string linea;
    ifstream file(filename);

    if (file.is_open())
    {
        while (getline(file, linea))
        {
            Point3D punto;
            stringstream ss(linea);
            string valore;
            if (getline(ss, valore, ','))
            {
                punto.x = stod(valore);
            }
            if (getline(ss, valore, ','))
            {
                punto.y = stod(valore);
            }
            if (getline(ss, valore))
            {
                punto.z = stod(valore);
            }
            punti.push_back(punto);
        }

        file.close();
    }
    return punti;
}

vector<Triangle> readTriangles(string filename, vector<Point3D> punti)
{
    vector<Triangle> triangoli;
    string linea;
    ifstream file(filename);

    if (file.is_open())
    {
        while (getline(file, linea))
        {
            Triangle t;
            stringstream ss(linea);
            string valore;

            int index = 0;

            if (getline(ss, valore, ','))
            {
                index = (int)stod(valore);
                t.p1 = punti[index];
            }
            if (getline(ss, valore, ','))
            {
                index = (int)stod(valore);
                t.p2 = punti[index];
            }
            if (getline(ss, valore))
            {
                index = (int)stod(valore);
                t.p3 = punti[index];
            }
            triangoli.push_back(t);
        }

        file.close();
    }
    return triangoli;
}

int main()
{
    vector<Point3D> punti = readPoints("verts.csv");
    vector<Triangle> triangoli = readTriangles("meshes.csv", punti);

	int Np = punti.size();
	int Nt = triangoli.size();

	Point3D rayOrigin = {0.0, 0.0, 0.0};
	
	Point3D *h_punti = new Point3D[Np];
	for(int i = 0; i < Np; i++){
		h_punti[i] = punti[i];
	}
	
	Triangle *h_triangoli = new Triangle[Nt];
	for(int i = 0; i < Nt; i++){
		h_triangoli[i] = triangoli[i];
	}
	
	bool *result = new bool[Np];
	
	Point3D *d_points;
	Triangle *d_triangles;
	bool *d_result;
	
	// DEVICE MEMORY ALLOCATION
	hipMalloc(&d_points, Np * sizeof(Point3D) );
	hipMalloc(&d_triangles, Nt * sizeof(Triangle) );
	hipMalloc(&d_result, Np * sizeof(bool) );
	
	// COPY DATA FROM HOST TO DEVIE
	hipMemcpy( d_points, h_punti, Np * sizeof(Point3D), hipMemcpyHostToDevice);
	hipMemcpy( d_triangles, h_triangoli, Nt * sizeof(Triangle), hipMemcpyHostToDevice);
	
	// DEVICE INIT
	dim3 DimGrid(Np/256, 1, 1);
	if (Np % 256)
		DimGrid.x++;
	dim3 DimBlock(256, 1, 1);
	
	// DEVICE EXECUTION
	rayIntersectsAnyTrianglesKernel<<<DimGrid, DimBlock>>>(rayOrigin, d_points, d_triangles, Nt, d_result);
	
	// COPY DATA FROM DEVICE TO HOST
	hipMemcpy(result, d_result, Np * sizeof(bool), hipMemcpyDeviceToHost);
	
	// DEVICE MEMORY DEALLOCATION
	hipFree( d_points );
	hipFree( d_triangles );
	hipFree( d_result );
	
	// PRINT RESULT
	ofstream oFile("out.txt");
	for(int i = 0; i < Np; i++) {
		oFile << result[i] << endl;
	}	
    oFile.close();
    return 0;
}